// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include <hip/hip_runtime.h>
#include "./cuda_runtime.h"

#define clip(x, a, b) x >= a ? (x < b ? x : b - 1) : a;

const int INTER_RESIZE_COEF_BITS = 11;
const int INTER_RESIZE_COEF_SCALE = 1 << INTER_RESIZE_COEF_BITS;

__global__ void resizeCudaKernel(const float* input,
                                 float* output,
                                 const int inputWidth,
                                 const int inputHeight,
                                 const int outputWidth,
                                 const int outputHeight,
                                 const int inputChannels) {
  // 2D Index of current thread
  const int dx = blockIdx.x * blockDim.x + threadIdx.x;
  const int dy = blockIdx.y * blockDim.y + threadIdx.y;

  if ((dx < outputWidth) && (dy < outputHeight)) {
    if (inputChannels == 1) {  // grayscale image
      // TODO(Zelda): support grayscale
    } else if (inputChannels == 3) {  // RGB image
      double scale_x = static_cast<double>(inputWidth / outputWidth);
      double scale_y = static_cast<double>(inputHeight / outputHeight);

      int xmax = outputWidth;

      float fx = static_cast<float>((dx + 0.5) * scale_x - 0.5);
      int sx = floorf(fx);
      fx = fx - sx;

      int isx1 = sx;
      if (isx1 < 0) {
        fx = 0.0;
        isx1 = 0;
      }
      if (isx1 >= (inputWidth - 1)) {
        xmax = ::min(xmax, dx);
        fx = 0;
        isx1 = inputWidth - 1;
      }

      float2 cbufx;
      cbufx.x = (1.f - fx);
      cbufx.y = fx;

      float fy = static_cast<float>((dy + 0.5) * scale_y - 0.5);
      int sy = floorf(fy);
      fy = fy - sy;

      int isy1 = clip(sy + 0, 0, inputHeight);
      int isy2 = clip(sy + 1, 0, inputHeight);

      float2 cbufy;
      cbufy.x = (1.f - fy);
      cbufy.y = fy;

      int isx2 = isx1 + 1;

      float3 d0;

      float3 s11 =
          make_float3(input[(isy1 * inputWidth + isx1) * inputChannels + 0],
                      input[(isy1 * inputWidth + isx1) * inputChannels + 1],
                      input[(isy1 * inputWidth + isx1) * inputChannels + 2]);
      float3 s12 =
          make_float3(input[(isy1 * inputWidth + isx2) * inputChannels + 0],
                      input[(isy1 * inputWidth + isx2) * inputChannels + 1],
                      input[(isy1 * inputWidth + isx2) * inputChannels + 2]);
      float3 s21 =
          make_float3(input[(isy2 * inputWidth + isx1) * inputChannels + 0],
                      input[(isy2 * inputWidth + isx1) * inputChannels + 1],
                      input[(isy2 * inputWidth + isx1) * inputChannels + 2]);
      float3 s22 =
          make_float3(input[(isy2 * inputWidth + isx2) * inputChannels + 0],
                      input[(isy2 * inputWidth + isx2) * inputChannels + 1],
                      input[(isy2 * inputWidth + isx2) * inputChannels + 2]);

      float h_rst00, h_rst01;
      // B
      if (dx > xmax - 1) {
        h_rst00 = s11.x;
        h_rst01 = s21.x;
      } else {
        h_rst00 = s11.x * cbufx.x + s12.x * cbufx.y;
        h_rst01 = s21.x * cbufx.x + s22.x * cbufx.y;
      }
      d0.x = h_rst00 * cbufy.x + h_rst01 * cbufy.y;

      // G
      if (dx > xmax - 1) {
        h_rst00 = s11.y;
        h_rst01 = s21.y;
      } else {
        h_rst00 = s11.y * cbufx.x + s12.y * cbufx.y;
        h_rst01 = s21.y * cbufx.x + s22.y * cbufx.y;
      }
      d0.y = h_rst00 * cbufy.x + h_rst01 * cbufy.y;
      // R
      if (dx > xmax - 1) {
        h_rst00 = s11.z;
        h_rst01 = s21.z;
      } else {
        h_rst00 = s11.z * cbufx.x + s12.z * cbufx.y;
        h_rst01 = s21.z * cbufx.x + s22.z * cbufx.y;
      }
      d0.z = h_rst00 * cbufy.x + h_rst01 * cbufy.y;

      output[(dy * outputWidth + dx) * 3 + 0] = (d0.x);  // R
      output[(dy * outputWidth + dx) * 3 + 1] = (d0.y);  // G
      output[(dy * outputWidth + dx) * 3 + 2] = (d0.z);  // B
    } else {
      // TODO(Zelda): support alpha channel
    }
  }
}

__global__ void resizeCudaKernel_fixpt(const float* input,
                                       float* output,
                                       const int inputWidth,
                                       const int inputHeight,
                                       const int outputWidth,
                                       const int outputHeight,
                                       const int inputChannels) {
  // 2D Index of current thread
  const int dx = blockIdx.x * blockDim.x + threadIdx.x;
  const int dy = blockIdx.y * blockDim.y + threadIdx.y;

  if ((dx < outputWidth) && (dy < outputHeight)) {
    if (inputChannels == 1) {  // grayscale image
      // TODO(Zelda): support grayscale
    } else if (inputChannels == 3) {  // RGB image
      double scale_x = static_cast<double>(inputWidth / outputWidth);
      double scale_y = static_cast<double>(inputHeight / outputHeight);

      int xmax = outputWidth;

      float fx = static_cast<float>((dx + 0.5) * scale_x - 0.5);
      int sx = floorf(fx);
      fx = fx - sx;

      int isx1 = sx;
      if (isx1 < 0) {
        fx = 0.0;
        isx1 = 0;
      }
      if (isx1 >= (inputWidth - 1)) {
        xmax = ::min(xmax, dx);
        fx = 0;
        isx1 = inputWidth - 1;
      }

      short2 cbufx;
      cbufx.x = lrintf((1.f - fx) * INTER_RESIZE_COEF_SCALE);
      cbufx.y = lrintf(fx * INTER_RESIZE_COEF_SCALE);

      float fy = static_cast<float>((dy + 0.5) * scale_y - 0.5);
      int sy = floorf(fy);
      fy = fy - sy;

      int isy1 = clip(sy + 0, 0, inputHeight);
      int isy2 = clip(sy + 1, 0, inputHeight);

      short2 cbufy;
      cbufy.x = lrintf((1.f - fy) * INTER_RESIZE_COEF_SCALE);
      cbufy.y = lrintf(fy * INTER_RESIZE_COEF_SCALE);

      int isx2 = isx1 + 1;

      uchar3 d0;

      int3 s11 =
          make_int3(input[(isy1 * inputWidth + isx1) * inputChannels + 0],
                    input[(isy1 * inputWidth + isx1) * inputChannels + 1],
                    input[(isy1 * inputWidth + isx1) * inputChannels + 2]);
      int3 s12 =
          make_int3(input[(isy1 * inputWidth + isx2) * inputChannels + 0],
                    input[(isy1 * inputWidth + isx2) * inputChannels + 1],
                    input[(isy1 * inputWidth + isx2) * inputChannels + 2]);
      int3 s21 =
          make_int3(input[(isy2 * inputWidth + isx1) * inputChannels + 0],
                    input[(isy2 * inputWidth + isx1) * inputChannels + 1],
                    input[(isy2 * inputWidth + isx1) * inputChannels + 2]);
      int3 s22 =
          make_int3(input[(isy2 * inputWidth + isx2) * inputChannels + 0],
                    input[(isy2 * inputWidth + isx2) * inputChannels + 1],
                    input[(isy2 * inputWidth + isx2) * inputChannels + 2]);

      int h_rst00, h_rst01;
      // B
      if (dx > xmax - 1) {
        h_rst00 = s11.x * INTER_RESIZE_COEF_SCALE;
        h_rst01 = s21.x * INTER_RESIZE_COEF_SCALE;
      } else {
        h_rst00 = s11.x * cbufx.x + s12.x * cbufx.y;
        h_rst01 = s21.x * cbufx.x + s22.x * cbufx.y;
      }
      d0.x = (unsigned char)((((cbufy.x * (h_rst00 >> 4)) >> 16) +
                              ((cbufy.y * (h_rst01 >> 4)) >> 16) + 2) >>
                             2);

      // G
      if (dx > xmax - 1) {
        h_rst00 = s11.y * INTER_RESIZE_COEF_SCALE;
        h_rst01 = s21.y * INTER_RESIZE_COEF_SCALE;
      } else {
        h_rst00 = s11.y * cbufx.x + s12.y * cbufx.y;
        h_rst01 = s21.y * cbufx.x + s22.y * cbufx.y;
      }
      d0.y = (unsigned char)((((cbufy.x * (h_rst00 >> 4)) >> 16) +
                              ((cbufy.y * (h_rst01 >> 4)) >> 16) + 2) >>
                             2);
      // R
      if (dx > xmax - 1) {
        h_rst00 = s11.z * INTER_RESIZE_COEF_SCALE;
        h_rst01 = s21.z * INTER_RESIZE_COEF_SCALE;
      } else {
        h_rst00 = s11.z * cbufx.x + s12.z * cbufx.y;
        h_rst01 = s21.z * cbufx.x + s22.z * cbufx.y;
      }
      d0.z = (unsigned char)((((cbufy.x * (h_rst00 >> 4)) >> 16) +
                              ((cbufy.y * (h_rst01 >> 4)) >> 16) + 2) >>
                             2);

      output[(dy * outputWidth + dx) * 3 + 0] = (d0.x);  // R
      output[(dy * outputWidth + dx) * 3 + 1] = (d0.y);  // G
      output[(dy * outputWidth + dx) * 3 + 2] = (d0.z);  // B
    } else {
      // TODO(Zelda): support alpha channel
    }
  }
}

extern "C" hipError_t resize_linear(const float* input,
                                     float* output,
                                     const int inputWidth,
                                     const int inputHeight,
                                     const int outputWidth,
                                     const int outputHeight,
                                     const int inputChannels,
                                     const bool use_fixed_point) {
  // Specify a reasonable block size
  const dim3 block(16, 16);

  // Calculate grid size to cover the whole image
  const dim3 grid((outputWidth + block.x - 1) / block.x,
                  (outputHeight + block.y - 1) / block.y);

  // Launch the size conversion kernel
  if (use_fixed_point) {
    resizeCudaKernel_fixpt<<<grid, block>>>(input,
                                            output,
                                            inputWidth,
                                            inputHeight,
                                            outputWidth,
                                            outputHeight,
                                            inputChannels);
  } else {
    resizeCudaKernel<<<grid, block>>>(input,
                                      output,
                                      inputWidth,
                                      inputHeight,
                                      outputWidth,
                                      outputHeight,
                                      inputChannels);
  }

  // Synchronize to check for any kernel launch errors
  return hipDeviceSynchronize();
}
